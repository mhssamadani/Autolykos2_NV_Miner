#include "hip/hip_runtime.h"
// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolykos puzzle cycle

*******************************************************************************/

#ifdef _WIN32
#ifndef WIN32_LEAN_AND_MEAN
#define WIN32_LEAN_AND_MEAN
#endif
#endif

#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/easylogging++.h"
#include "../include/jsmn.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/processing.h"
#include "../include/reduction.h"
#include "../include/request.h"
#include "../include/httpapi.h"
#include "../include/queue.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <iostream>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <atomic>
#include <chrono>
#include <mutex>
#include <thread>
#include <vector>
#include <random>

#ifdef _WIN32
#include <io.h>
#define R_OK 4       
#define W_OK 2       
#define F_OK 0       
#define access _access
#else
#include <unistd.h>
#endif

INITIALIZE_EASYLOGGINGPP

using namespace std::chrono;

std::atomic<int> end_jobs(0);

void SenderThread(info_t * info, BlockQueue<MinerShare>* shQueue)
{
	el::Helpers::setThreadName("sender thread");
    while(true)
    {
		MinerShare share = shQueue->get();
		char logstr[2048];

			LOG(INFO) << "Some GPU found and trying to POST a share: " ;
			PostPuzzleSolution(info->to, (uint8_t*)&share.nonce);
        

    }


}

////////////////////////////////////////////////////////////////////////////////
//  Miner thread cycle
////////////////////////////////////////////////////////////////////////////////
void MinerThread(const int totalGPUCards, int deviceId, info_t * info, std::vector<double>* hashrates, std::vector<int>* tstamps, BlockQueue<MinerShare>* shQueue)
{
    CUDA_CALL(hipSetDevice(deviceId));
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    char threadName[20];
    sprintf(threadName, "GPU %i miner", deviceId);
    el::Helpers::setThreadName(threadName);    

    state_t state = STATE_KEYGEN;
    char logstr[1000];

    //========================================================================//
    //  Host memory allocation
    //========================================================================//
    // CURL http request
    json_t request(0, REQ_LEN);

    // hash context
    // (212 + 4) bytes
    ctx_t ctx_h;

    // autolykos variables
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t nonce[NONCE_SIZE_8];

    char to[MAX_URL_SIZE];
    int keepPrehash = 0;

    // thread info variables
    uint_t blockId = 0;
    milliseconds start; 
    
    //========================================================================//
    //  Copy from global to thread local data
    //========================================================================//
    info->info_mutex.lock();

    memcpy(mes_h, info->mes, NUM_SIZE_8);
    memcpy(bound_h, info->bound, NUM_SIZE_8);
    memcpy(to, info->to, MAX_URL_SIZE * sizeof(char));
    // blockId = info->blockId.load();
    keepPrehash = info->keepPrehash;
    
    info->info_mutex.unlock();
    
    //========================================================================//
    //  Check GPU memory
    //========================================================================//
    size_t freeMem;
    size_t totalMem;

    CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
    
    if (freeMem < MIN_FREE_MEMORY)
    {
        LOG(ERROR) << "Not enough GPU memory for mining,"
            << " minimum 2.8 GiB needed";

        return;
    }

    keepPrehash = 0;

    //========================================================================//
    //  Device memory allocation
    //========================================================================//
    LOG(INFO) << "GPU " << deviceId << " allocating memory";

    // height for puzzle
    uint32_t * height_d;
    CUDA_CALL(hipMalloc(&height_d, HEIGHT_SIZE));

    // boundary for puzzle
    uint32_t * bound_d;
    // (2 * PK_SIZE_8 + 2 + 4 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    CUDA_CALL(hipMalloc(&bound_d, NUM_SIZE_8 + DATA_SIZE_8));
    // data: pk || mes || w || padding || x || sk || ctx
    uint32_t * data_d = bound_d + NUM_SIZE_32;

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc(&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // place to handle result of the puzzle
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc(&indices_d, MAX_SOLS*sizeof(uint32_t)));

    // place to handle nonce if solution is found
    uint32_t indices_h[MAX_SOLS];
    
    uint32_t * count_d;

    CUDA_CALL(hipMalloc(&count_d,sizeof(uint32_t)));

    CUDA_CALL(hipMemset(count_d,0,sizeof(uint32_t)));

    
    CUDA_CALL(hipMemset(
        indices_d, 0, sizeof(uint32_t)*MAX_SOLS
    ));

    // unfinalized hash contexts
    // if keepPrehash == true // N_LEN * 80 bytes // 5 GiB
    uctx_t * uctxs_d = NULL;
    if(info->AlgVer == 2)
        keepPrehash = false;
    if (keepPrehash)
    {
        CUDA_CALL(hipMalloc(&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)));
    }


    //========================================================================//
    //  Autolykos puzzle cycle
    //========================================================================//
    uint32_t ind = 0;
    uint64_t base = 0;
	uint64_t EndNonce = 0;

    uint32_t height = 0;



    int cntCycles = 0;
    int NCycles = 50;

    // wait for the very first block to come before starting
    while (info->blockId.load() == 0) {}

    start = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    do
    {
        ++cntCycles;

        if (!(cntCycles % NCycles))
        {
            milliseconds timediff
                = duration_cast<milliseconds>(
                    system_clock::now().time_since_epoch()
                ) - start;
            
            // change avg hashrate in global memory

            (*hashrates)[deviceId] = (double)NONCES_PER_ITER * (double)NCycles
                / ((double)1000 * timediff.count());
             
	    
            start = duration_cast<milliseconds>(
                system_clock::now().time_since_epoch()
            );

            (*tstamps)[deviceId] = start.count();
        }
    
        // if solution was found by this thread wait for new block to come 
        if (state == STATE_KEYGEN)
        {
            while (info->blockId.load() == blockId) {}

            state = STATE_CONTINUE;
        }

        uint_t controlId = info->blockId.load();
        
        if (blockId != controlId)
        {
            // if info->blockId changed
            // read new message and bound to thread-local mem
            info->info_mutex.lock();

            memcpy(mes_h, info->mes, NUM_SIZE_8);
            memcpy(bound_h, info->bound, NUM_SIZE_8);


			//divide nonces between gpus
			memcpy(&EndNonce, info->extraNonceEnd, NONCE_SIZE_8);
			memcpy(&base, info->extraNonceStart, NONCE_SIZE_8);
			uint64_t nonceChunk = 1 + (EndNonce - base) / totalGPUCards;
			base = *((uint64_t *)info->extraNonceStart) + deviceId * nonceChunk;
            EndNonce = base + nonceChunk;
            
            
            memcpy(&height,info->Hblock, HEIGHT_SIZE);

            info->info_mutex.unlock();

            LOG(INFO) << "GPU " << deviceId << " read new block data";
            blockId = controlId;
            

            VLOG(1) << "Generated new keypair,"
                << " copying new data in device memory now";

            // copy boundary
            CUDA_CALL(hipMemcpy(
                bound_d, bound_h, NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy message
            CUDA_CALL(hipMemcpy(
                ((uint8_t *)data_d + PK_SIZE_8), mes_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));



            VLOG(1) << "Starting prehashing with new block data";
            Prehash(keepPrehash, data_d, uctxs_d, hashes_d,height,info->AlgVer);
            
            // calculate unfinalized hash of message
            VLOG(1) << "Starting InitMining";
            InitMining(&ctx_h, (uint32_t *)mes_h, NUM_SIZE_8);
            
            CUDA_CALL(hipDeviceSynchronize());
			LOG(INFO) << "GPU " << deviceId << " started";
            
            // copy context
            CUDA_CALL(hipMemcpy(
                data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32, &ctx_h,
                sizeof(ctx_t), hipMemcpyHostToDevice
            ));

            state = STATE_CONTINUE;
        }

        //LOG(INFO) << "Starting main BlockMining procedure";

        // calculate solution candidates

            // copy message
            CUDA_CALL(hipMemcpy(
                ((uint8_t *)data_d), mes_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));

            
            CUDA_CALL(hipMemcpy(
                ((uint8_t *)data_d)+ NUM_SIZE_8, &ctx_h, sizeof(ctx_t),
                hipMemcpyHostToDevice
            ));

                        


	int threads = THREADS_PER_ITER;
	uint64_t check = base + threads;
	if (check > EndNonce)
	{
		threads = EndNonce - base;
	}
	if (threads <= 0)
	{
        LOG(INFO) << " negative threads, ( base: " << base << " , endNonce: " << EndNonce << " ) ";
    }
    else
    {
            BlockMining<<<1 + (threads - 1) / BLOCK_DIM, BLOCK_DIM>>>(
                bound_d, data_d, base,height, hashes_d, indices_d , count_d
            );
    }
        VLOG(1) << "Trying to find solution";

        // restart iteration if new block was found
        if (blockId != info->blockId.load()) { continue; }


		CUDA_CALL(hipMemcpy(
            indices_h, indices_d, MAX_SOLS*sizeof(uint32_t),
            hipMemcpyDeviceToHost
        ));
		
		//exit(0);

        // solution found
        if (indices_h[0])
        {
            
            
			int i = 0;
			while (indices_h[i] && (i < 16/*MAX_SOLS*/))
			{

				*((uint64_t *)nonce) = base + indices_h[i] - 1;
				uint64_t endNonceT;
				memcpy(&endNonceT , info->extraNonceEnd , sizeof(uint64_t));
				if ( (*((uint64_t *)nonce)) <= endNonceT )
				{

                    MinerShare share(*((uint64_t *)nonce));
                    shQueue->put(share);


                    if (!info->stratumMode)
                    {
                        state = STATE_KEYGEN;
                        //end_jobs.fetch_add(1, std::memory_order_relaxed);
                        break;

                    }

                }
		else
		{
			//LOG(INFO) << "nonce greater than end nonce, nonce: " << *((uint64_t *)nonce) << " endNonce:  " << endNonceT;
		}
		i++;
	}

            memset(indices_h,0,MAX_SOLS*sizeof(uint32_t));
            CUDA_CALL(hipMemset(
                indices_d, 0, sizeof(uint32_t)
            ));
  			CUDA_CALL(hipMemset(count_d,0,sizeof(uint32_t)));
		
        }
       base += NONCES_PER_ITER;
       if (base > EndNonce) 	//end work
       {
           state = STATE_KEYGEN;
           end_jobs.fetch_add(1, std::memory_order_relaxed);
       }

    }
    while (1);
}

////////////////////////////////////////////////////////////////////////////////
//  Main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    //========================================================================//
    //  Setup log
    //========================================================================//
    START_EASYLOGGINGPP(argc, argv);

    el::Loggers::reconfigureAllLoggers(
        el::ConfigurationType::Format, "%datetime %level [%thread] %msg"
    );

    el::Helpers::setThreadName("main thread");

    char logstr[1000];


    //========================================================================//
    //  Check GPU availability
    //========================================================================//
    int deviceCount;
    int status = EXIT_SUCCESS;

    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        LOG(ERROR) << "Error checking GPU";
        return EXIT_FAILURE;
    }

    LOG(INFO) << "Using " << deviceCount << " GPU devices";

    //========================================================================//
    //  Read configuration file
    //========================================================================//
    char confName[14] = "./config.json";
    char * fileName = (argc == 1)? confName: argv[1];
    char from[MAX_URL_SIZE];
    info_t info;
    info.blockId = 0;
    info.keepPrehash = 0;
    
    BlockQueue<MinerShare> solQueue;


    LOG(INFO) << "Using configuration file " << fileName;

    // check access to config file
    if (access(fileName, F_OK) == -1)
    {
        LOG(ERROR) << "Configuration file " << fileName << " is not found";
        return EXIT_FAILURE;
    }

    // read configuration from file
    status = ReadConfig(
        fileName, from, info.to, info.endJob
     );

    if (status == EXIT_FAILURE) { return EXIT_FAILURE; }

    LOG(INFO) << "Block getting URL:\n   " << from;
    LOG(INFO) << "Solution posting URL:\n   " << info.to;


    //========================================================================//
    //  Setup CURL
    //========================================================================//
    // CURL http request
    json_t request(0, REQ_LEN);

    // CURL init
    PERSISTENT_CALL_STATUS(curl_global_init(CURL_GLOBAL_ALL), CURLE_OK);
    

    //========================================================================//
    //  Fork miner threads
    //========================================================================//
    std::vector<std::thread> miners(deviceCount);
    std::vector<double> hashrates(deviceCount);
    std::vector<int> lastTimestamps(deviceCount);
    std::vector<int> timestamps(deviceCount);
    
    // PCI bus and device IDs
    std::vector<std::pair<int,int>> devinfos(deviceCount);
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t props;
        if(hipGetDeviceProperties(&props, i) == hipSuccess)
        {
            devinfos[i] = std::make_pair(props.pciBusID, props.pciDeviceID);
        }
        miners[i] = std::thread(MinerThread,deviceCount, i, &info, &hashrates, &timestamps, &solQueue);
        hashrates[i] = 0;
        lastTimestamps[i] = 1;
        timestamps[i] = 0;
    }


    // get first block 
    status = EXIT_FAILURE;
    while(status != EXIT_SUCCESS)
    {
        status = GetLatestBlock(from, &request, &info, 0);
        std::this_thread::sleep_for(std::chrono::milliseconds(800));
        if(status != EXIT_SUCCESS)
        {
            LOG(INFO) << "Waiting for block data to be published by node...";
        }
    }
    std::thread solSender(SenderThread, &info, &solQueue);
    std::thread httpApi = std::thread(HttpApiThread,&hashrates,&devinfos);    

    //========================================================================//
    //  Main thread get-block cycle
    //========================================================================//
    uint_t curlcnt = 0;
    const uint_t curltimes = 500;

    milliseconds ms = milliseconds::zero(); 
    


    // bomb node with HTTP with 10ms intervals, if new block came 
    // signal miners with blockId
    while (1)
    {
        milliseconds start = duration_cast<milliseconds>(
            system_clock::now().time_since_epoch()
        );
        
        // get latest block
        status = GetLatestBlock(from, &request, &info, 0);
        
        if (status != EXIT_SUCCESS) { LOG(INFO) << "Getting block error"; }

        ms += duration_cast<milliseconds>(
            system_clock::now().time_since_epoch()
        ) - start;

        ++curlcnt;

        if (!(curlcnt % curltimes))
        {
            LOG(INFO) << "Average curling time "
                << ms.count() / (double)curltimes << " ms";
            LOG(INFO) << "Current block candidate: " << request.ptr;
            ms = milliseconds::zero();
            std::stringstream hrBuffer;
            hrBuffer << "Average hashrates: ";
            double totalHr = 0;
            for(int i = 0; i < deviceCount; ++i)
            {
                // check if miner thread is updating hashrate, e.g. alive
                if(!(curlcnt % (5*curltimes)))
                {
                    if(lastTimestamps[i] == timestamps[i])
                    {
                        hashrates[i] = 0;

		    }
                    lastTimestamps[i] = timestamps[i];
                }
                hrBuffer << "GPU" << i << " " << hashrates[i] << " MH/s ";
                totalHr += hashrates[i];
                
            }
            hrBuffer << "Total " << totalHr << " MH/s ";
            LOG(INFO) << hrBuffer.str();
        }

        std::this_thread::sleep_for(std::chrono::milliseconds(60));

        int completeMiners = end_jobs.load();
		if (completeMiners >= deviceCount)
		{
			end_jobs.store(0);
			JobCompleted(info.endJob);
		}
    }    

    return EXIT_SUCCESS;
}

// autolykos.cu

